#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <array>
#include <xyz_autodiff/dense_matrix.cuh>
#include <xyz_autodiff/diagonal_matrix_view.cuh>
#include <xyz_autodiff/variable.cuh>
#include <xyz_autodiff/util/cuda_unique_ptr.cuh>

using namespace xyz_autodiff;

// テスト用バッファ構造体（シンプル版）
template <typename T>
struct MatrixTransposeTestBuffers {
    T data[10];    // 入力データ用
    T result[20];  // 結果格納用
};

// DenseMatrix transpose テスト用CUDAカーネル
template <typename T>
__global__ void test_matrix_view_transpose_kernel(T* result) {
    // 2x3行列を作成
    DenseMatrix<T, 2, 3> matrix;
    
    // データ設定
    matrix(0, 0) = static_cast<T>(1.0);  // [1, 2, 3]
    matrix(0, 1) = static_cast<T>(2.0);  // [4, 5, 6]
    matrix(0, 2) = static_cast<T>(3.0);
    matrix(1, 0) = static_cast<T>(4.0);
    matrix(1, 1) = static_cast<T>(5.0);
    matrix(1, 2) = static_cast<T>(6.0);
    
    // transpose操作
    auto transposed = matrix.transpose();
    
    // transposed は 3x2 行列
    // [1, 4]
    // [2, 5] 
    // [3, 6]
    result[0] = transposed(0, 0);  // 1
    result[1] = transposed(0, 1);  // 4
    result[2] = transposed(1, 0);  // 2
    result[3] = transposed(1, 1);  // 5
    result[4] = transposed(2, 0);  // 3
    result[5] = transposed(2, 1);  // 6
    
    // 二重transpose（元に戻る）
    auto double_transposed = transposed.transpose();
    result[6] = double_transposed(0, 0);  // 1
    result[7] = double_transposed(1, 2);  // 6
}

// DenseMatrix transpose テスト用CUDAカーネル
template <typename T>
__global__ void test_dense_matrix_transpose_kernel(T* result) {
    DenseMatrix<T, 2, 3> matrix;
    
    // データ設定
    matrix(0, 0) = static_cast<T>(1.0);
    matrix(0, 1) = static_cast<T>(2.0);
    matrix(0, 2) = static_cast<T>(3.0);
    matrix(1, 0) = static_cast<T>(4.0);
    matrix(1, 1) = static_cast<T>(5.0);
    matrix(1, 2) = static_cast<T>(6.0);
    
    // transpose操作
    auto transposed = matrix.transpose();
    
    // 結果確認
    result[0] = transposed(0, 0);  // 1
    result[1] = transposed(0, 1);  // 4
    result[2] = transposed(1, 0);  // 2
    result[3] = transposed(1, 1);  // 5
    result[4] = transposed(2, 0);  // 3
    result[5] = transposed(2, 1);  // 6
}

// DiagonalMatrix transpose テスト用CUDAカーネル
template <typename T>
__global__ void test_diagonal_matrix_transpose_kernel(T* data, T* result) {
    VariableRef<3, T> var(data, data + 3);  // data後半を勾配用に使用
    
    // 対角要素設定
    var[0] = static_cast<T>(1.0);
    var[1] = static_cast<T>(2.0);
    var[2] = static_cast<T>(3.0);
    
    DiagonalMatrixView<T, 3, VariableRef<3, T>> diag_view(var);
    
    // transpose操作（対角行列なので変わらない）
    auto transposed = diag_view.transpose();
    
    // 結果確認
    result[0] = transposed(0, 0);  // 1
    result[1] = transposed(1, 1);  // 2  
    result[2] = transposed(2, 2);  // 3
    result[3] = transposed(0, 1);  // 0（非対角要素）
    result[4] = transposed(1, 0);  // 0（非対角要素）
}

class MatrixTransposeTest : public ::testing::Test {
protected:
    void SetUp() override {
        hipError_t err = hipSetDevice(0);
        if (err != hipSuccess) {
            GTEST_SKIP() << "CUDA device not available";
        }
    }
};

TEST_F(MatrixTransposeTest, MatrixViewTranspose) {
    using T = float;
    
    auto device_buffers = makeCudaUnique<MatrixTransposeTestBuffers<T>>();
    ASSERT_NE(device_buffers, nullptr);
    
    test_matrix_view_transpose_kernel<T><<<1, 1>>>(device_buffers.get()->result);
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    
    MatrixTransposeTestBuffers<T> host_buffers;
    ASSERT_EQ(hipMemcpy(&host_buffers, device_buffers.get(), sizeof(MatrixTransposeTestBuffers<T>), hipMemcpyDeviceToHost), hipSuccess);
    
    // transpose結果の検証
    EXPECT_FLOAT_EQ(host_buffers.result[0], 1.0f);  // (0,0) = 1
    EXPECT_FLOAT_EQ(host_buffers.result[1], 4.0f);  // (0,1) = 4
    EXPECT_FLOAT_EQ(host_buffers.result[2], 2.0f);  // (1,0) = 2
    EXPECT_FLOAT_EQ(host_buffers.result[3], 5.0f);  // (1,1) = 5
    EXPECT_FLOAT_EQ(host_buffers.result[4], 3.0f);  // (2,0) = 3
    EXPECT_FLOAT_EQ(host_buffers.result[5], 6.0f);  // (2,1) = 6
    
    // 二重transpose結果の検証
    EXPECT_FLOAT_EQ(host_buffers.result[6], 1.0f);  // (0,0) = 1
    EXPECT_FLOAT_EQ(host_buffers.result[7], 6.0f);  // (1,2) = 6
}

TEST_F(MatrixTransposeTest, DenseMatrixTranspose) {
    using T = float;
    
    auto device_buffers = makeCudaUnique<MatrixTransposeTestBuffers<T>>();
    ASSERT_NE(device_buffers, nullptr);
    
    test_dense_matrix_transpose_kernel<T><<<1, 1>>>(device_buffers.get()->result);
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    
    MatrixTransposeTestBuffers<T> host_buffers;
    ASSERT_EQ(hipMemcpy(&host_buffers, device_buffers.get(), sizeof(MatrixTransposeTestBuffers<T>), hipMemcpyDeviceToHost), hipSuccess);
    
    // transpose結果の検証
    EXPECT_FLOAT_EQ(host_buffers.result[0], 1.0f);  // (0,0) = 1
    EXPECT_FLOAT_EQ(host_buffers.result[1], 4.0f);  // (0,1) = 4
    EXPECT_FLOAT_EQ(host_buffers.result[2], 2.0f);  // (1,0) = 2
    EXPECT_FLOAT_EQ(host_buffers.result[3], 5.0f);  // (1,1) = 5
    EXPECT_FLOAT_EQ(host_buffers.result[4], 3.0f);  // (2,0) = 3
    EXPECT_FLOAT_EQ(host_buffers.result[5], 6.0f);  // (2,1) = 6
}

TEST_F(MatrixTransposeTest, DiagonalMatrixTranspose) {
    using T = float;
    
    auto device_buffers = makeCudaUnique<MatrixTransposeTestBuffers<T>>();
    ASSERT_NE(device_buffers, nullptr);
    
    test_diagonal_matrix_transpose_kernel<T><<<1, 1>>>(
        device_buffers.get()->data, device_buffers.get()->result);
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    
    MatrixTransposeTestBuffers<T> host_buffers;
    ASSERT_EQ(hipMemcpy(&host_buffers, device_buffers.get(), sizeof(MatrixTransposeTestBuffers<T>), hipMemcpyDeviceToHost), hipSuccess);
    
    // transpose結果の検証（対角行列なので変わらない）
    EXPECT_FLOAT_EQ(host_buffers.result[0], 1.0f);  // (0,0) = 1
    EXPECT_FLOAT_EQ(host_buffers.result[1], 2.0f);  // (1,1) = 2
    EXPECT_FLOAT_EQ(host_buffers.result[2], 3.0f);  // (2,2) = 3
    EXPECT_FLOAT_EQ(host_buffers.result[3], 0.0f);  // (0,1) = 0
    EXPECT_FLOAT_EQ(host_buffers.result[4], 0.0f);  // (1,0) = 0
}

// Concept チェックテスト
TEST_F(MatrixTransposeTest, ConceptCheck) {
    using DenseMat23 = DenseMatrix<float, 2, 3>;
    using DenseMat32 = DenseMatrix<float, 3, 2>;
    using DiagView3 = DiagonalMatrixView<float, 3, VariableRef<3, float>>;
    
    // MatrixViewConcept の要件チェック
    static_assert(MatrixViewConcept<DenseMat23>);
    static_assert(MatrixViewConcept<DenseMat32>);
    static_assert(MatrixViewConcept<DiagView3>);
    
    // サイズ情報のチェック
    static_assert(DenseMat23::rows == 2 && DenseMat23::cols == 3);
    static_assert(DenseMat32::rows == 3 && DenseMat32::cols == 2);
    static_assert(DiagView3::rows == 3 && DiagView3::cols == 3);
}

int main(int argc, char** argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}