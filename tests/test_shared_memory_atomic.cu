#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cmath>
#include <xyz_autodiff/variable.cuh>
#include <xyz_autodiff/util/cuda_unique_ptr.cuh>
#include <xyz_autodiff/variable_operators.cuh>

using namespace xyz_autodiff;

// テスト用バッファ構造体
struct SharedMemoryTestBuffers {
    double x_grad;
    double y_grad;
};

// マルチブロック用バッファ構造体
struct MultiBlockTestBuffers {
    double x_grads[10];  // 最大10ブロック分
    double y_grads[10];  // 最大10ブロック分
};

/**
 * Shared Memory AtomicAdd テスト
 * shared memory上の変数に対して複数スレッドが
 * atomicAddで勾配を加算することを確認
 */

__global__ void shared_memory_atomic_kernel(
    SharedMemoryTestBuffers* global_results,
    std::size_t threads_per_block) {
    
    // Shared memory allocation
    __shared__ double shared_x_data[1];
    __shared__ double shared_x_grad[1];
    __shared__ double shared_y_data[1];
    __shared__ double shared_y_grad[1];
    
    // Initialize shared memory (only thread 0)
    if (threadIdx.x == 0) {
        shared_x_data[0] = 5.0;
        shared_x_grad[0] = 0.0;
        shared_y_data[0] = 3.0;
        shared_y_grad[0] = 0.0;
    }
    
    __syncthreads(); // Ensure initialization is complete
    
    // Create VariableRef pointing to shared memory
    VariableRef<1, double> x_ref(shared_x_data, shared_x_grad);
    VariableRef<1, double> y_ref(shared_y_data, shared_y_grad);
    
    // Each thread adds gradient using atomicAdd on shared memory
    x_ref.add_grad(0, 1.0);  // atomicAdd on shared memory
    y_ref.add_grad(0, 2.0);  // atomicAdd on shared memory
    
    __syncthreads(); // Ensure all atomic operations are complete
    
    // Copy results back to global memory (only thread 0)
    if (threadIdx.x == 0) {
        global_results->x_grad = shared_x_grad[0];
        global_results->y_grad = shared_y_grad[0];
    }
}

class SharedMemoryAtomicTest : public ::testing::Test {
protected:
    void SetUp() override {
        hipError_t err = hipSetDevice(0);
        if (err != hipSuccess) {
            GTEST_SKIP() << "CUDA device not available";
        }
    }
};

TEST_F(SharedMemoryAtomicTest, SharedMemoryAtomicAddGradient) {
    const std::size_t THREADS_PER_BLOCK = 128;
    const std::size_t NUM_BLOCKS = 1; // Single block to test shared memory
    
    // Device memory for results
    auto device_results = makeCudaUnique<SharedMemoryTestBuffers>();
    ASSERT_NE(device_results, nullptr);
    
    // Initialize results to zero
    SharedMemoryTestBuffers zero_buffers = {0.0, 0.0};
    ASSERT_EQ(hipMemcpy(device_results.get(), &zero_buffers, sizeof(SharedMemoryTestBuffers), hipMemcpyHostToDevice), hipSuccess);
    
    // Launch kernel
    shared_memory_atomic_kernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(
        device_results.get(),
        THREADS_PER_BLOCK
    );
    
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    
    // Copy results back to host
    SharedMemoryTestBuffers host_results;
    ASSERT_EQ(hipMemcpy(&host_results, device_results.get(), sizeof(SharedMemoryTestBuffers), hipMemcpyDeviceToHost), hipSuccess);
    
    // Verify results
    // Each of THREADS_PER_BLOCK threads adds 1.0 to x_grad
    double expected_x_grad = static_cast<double>(THREADS_PER_BLOCK);
    // Each of THREADS_PER_BLOCK threads adds 2.0 to y_grad
    double expected_y_grad = static_cast<double>(THREADS_PER_BLOCK) * 2.0;
    
    EXPECT_NEAR(host_results.x_grad, expected_x_grad, 1e-10)
        << "x gradient should be " << expected_x_grad << " but got " << host_results.x_grad;
    EXPECT_NEAR(host_results.y_grad, expected_y_grad, 1e-10)
        << "y gradient should be " << expected_y_grad << " but got " << host_results.y_grad;
    
    // Success message
    std::cout << "SUCCESS: Shared memory atomicAdd works correctly!" << std::endl;
    std::cout << "x_grad: " << host_results.x_grad << " (expected: " << expected_x_grad << ")" << std::endl;
    std::cout << "y_grad: " << host_results.y_grad << " (expected: " << expected_y_grad << ")" << std::endl;
    std::cout << "AtomicAdd performed correctly on shared memory with " << THREADS_PER_BLOCK << " threads." << std::endl;
}

// Multi-block shared memory atomic kernel
__global__ void multi_block_shared_memory_kernel(MultiBlockTestBuffers* results, std::size_t threads_per_block) {
    // Shared memory allocation (per block)
    __shared__ double shared_x_data[1];
    __shared__ double shared_x_grad[1];
    __shared__ double shared_y_data[1];
    __shared__ double shared_y_grad[1];
    
    // Initialize shared memory (only thread 0 in each block)
    if (threadIdx.x == 0) {
        shared_x_data[0] = 5.0;
        shared_x_grad[0] = 0.0;
        shared_y_data[0] = 3.0;
        shared_y_grad[0] = 0.0;
    }
    
    __syncthreads();
    
    // Create VariableRef pointing to shared memory
    VariableRef<1, double> x_ref(shared_x_data, shared_x_grad);
    VariableRef<1, double> y_ref(shared_y_data, shared_y_grad);
    
    // Each thread adds gradient using atomicAdd on shared memory
    x_ref.add_grad(0, 1.0);
    y_ref.add_grad(0, 2.0);
    
    __syncthreads();
    
    // Copy results back to global memory (only thread 0 in each block)
    if (threadIdx.x == 0) {
        results->x_grads[blockIdx.x] = shared_x_grad[0];
        results->y_grads[blockIdx.x] = shared_y_grad[0];
    }
}

// Test with multiple blocks to verify shared memory isolation
TEST_F(SharedMemoryAtomicTest, MultiBlockSharedMemoryAtomic) {
    const std::size_t THREADS_PER_BLOCK = 64;
    const std::size_t NUM_BLOCKS = 4;
    
    // Device memory for results (one per block)
    auto device_results = makeCudaUnique<MultiBlockTestBuffers>();
    ASSERT_NE(device_results, nullptr);
    
    // Initialize all results to zero
    MultiBlockTestBuffers zero_results = {};
    ASSERT_EQ(hipMemcpy(device_results.get(), &zero_results, sizeof(MultiBlockTestBuffers), hipMemcpyHostToDevice), hipSuccess);
    
    // Launch kernel
    multi_block_shared_memory_kernel<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(
        device_results.get(),
        THREADS_PER_BLOCK
    );
    
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    
    // Copy results back to host
    MultiBlockTestBuffers host_results;
    ASSERT_EQ(hipMemcpy(&host_results, device_results.get(), sizeof(MultiBlockTestBuffers), hipMemcpyDeviceToHost), hipSuccess);
    
    // Verify results for each block
    double expected_x_grad = static_cast<double>(THREADS_PER_BLOCK);
    double expected_y_grad = static_cast<double>(THREADS_PER_BLOCK) * 2.0;
    
    for (std::size_t block = 0; block < NUM_BLOCKS; ++block) {
        EXPECT_NEAR(host_results.x_grads[block], expected_x_grad, 1e-10)
            << "Block " << block << " x gradient should be " << expected_x_grad << " but got " << host_results.x_grads[block];
        EXPECT_NEAR(host_results.y_grads[block], expected_y_grad, 1e-10)
            << "Block " << block << " y gradient should be " << expected_y_grad << " but got " << host_results.y_grads[block];
    }
    
    std::cout << "SUCCESS: Multi-block shared memory atomicAdd works correctly!" << std::endl;
    std::cout << "Tested " << NUM_BLOCKS << " blocks with " << THREADS_PER_BLOCK << " threads each." << std::endl;
}

int main(int argc, char** argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}