#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <type_traits>
#include "../include/variable.cuh"
#include "../include/operation.cuh"
#include "../include/operations/add_logic.cuh"
#include "../include/util/cuda_unique_ptr.cuh"

using namespace xyz_autodiff;

// Operation テスト用のCUDAカーネル
template <typename T>
__global__ void test_operation_kernel(T* data1, T* grad1, T* data2, T* grad2, T* result) {
    // Variable作成
    Variable<T, 1> var1(data1, grad1);
    Variable<T, 1> var2(data2, grad2);
    
    // 値設定
    var1[0] = static_cast<T>(3.0);
    var2[0] = static_cast<T>(4.0);
    
    // make_add関数を使用してBinaryOperationを作成（内部でbufferを保持）
    auto op = make_add<1>(var1, var2);
    
    // forward計算
    op.forward();
    result[0] = op.output()[0];
    
    // 出力に単位勾配を設定してbackward計算
    op.output().grad(0) = static_cast<T>(1.0);
    
    // backward計算
    op.backward();
    
    // 勾配結果を保存
    result[1] = var1.grad(0);  // dL/dvar1
    result[2] = var2.grad(0);  // dL/dvar2
}

class OperationTest : public ::testing::Test {
protected:
    void SetUp() override {
        hipError_t err = hipSetDevice(0);
        if (err != hipSuccess) {
            GTEST_SKIP() << "CUDA device not available";
        }
    }
};

TEST_F(OperationTest, BasicAddition) {
    using T = float;
    
    // ホストメモリ
    std::vector<T> host_result(3, 0);
    
    // デバイスメモリ確保
    auto device_data1 = makeCudaUniqueArray<T>(1);
    auto device_grad1 = makeCudaUniqueArray<T>(1);
    auto device_data2 = makeCudaUniqueArray<T>(1);
    auto device_grad2 = makeCudaUniqueArray<T>(1);
    auto device_result = makeCudaUniqueArray<T>(3);
    
    ASSERT_NE(device_data1, nullptr);
    ASSERT_NE(device_grad1, nullptr);
    ASSERT_NE(device_data2, nullptr);
    ASSERT_NE(device_grad2, nullptr);
    ASSERT_NE(device_result, nullptr);
    
    // 勾配初期化
    T zero = 0.0f;
    ASSERT_EQ(hipMemcpy(device_grad1.get(), &zero, sizeof(T), hipMemcpyHostToDevice), hipSuccess);
    ASSERT_EQ(hipMemcpy(device_grad2.get(), &zero, sizeof(T), hipMemcpyHostToDevice), hipSuccess);
    
    // カーネル実行
    test_operation_kernel<T><<<1, 1>>>(
        device_data1.get(), device_grad1.get(),
        device_data2.get(), device_grad2.get(),
        device_result.get());
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    
    // 結果をホストにコピー
    ASSERT_EQ(hipMemcpy(host_result.data(), device_result.get(), 3 * sizeof(T), hipMemcpyDeviceToHost), hipSuccess);
    
    // 検証
    EXPECT_FLOAT_EQ(host_result[0], 7.0f);  // 3 + 4 = 7
    EXPECT_FLOAT_EQ(host_result[1], 1.0f);  // d(3+4)/d3 = 1
    EXPECT_FLOAT_EQ(host_result[2], 1.0f);  // d(3+4)/d4 = 1
}

TEST_F(OperationTest, ConceptCheck) {
    using Var1 = Variable<float, 1>;
    using AddLogicType = AddLogic<Var1, Var1, 1>;
    using AddOp = BinaryOperation<AddLogicType, Var1, Var1, 1>;
    
    // 型要件のチェック
    static_assert(std::is_same_v<AddOp::input1_type, Var1>);
    static_assert(std::is_same_v<AddOp::input2_type, Var1>);
    static_assert(std::is_same_v<AddOp::value_type, float>);
    static_assert(AddOp::output_size == 1);
}

int main(int argc, char** argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}