#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <type_traits>
#include <array>
#include "../include/variable.cuh"
#include "../include/operations/operation.cuh"
#include "../include/operations/add_logic.cuh"
#include "../include/util/cuda_unique_ptr.cuh"

using namespace xyz_autodiff;

// テスト用汎用バッファ構造体
template <typename T, std::size_t NumVariables, std::size_t VarSize = 1>
class TestBuffer {
public:
    // ホスト側データ
    std::array<std::array<T, VarSize>, NumVariables> host_data;
    std::array<std::array<T, VarSize>, NumVariables> host_grad;
    std::vector<T> host_result;
    
    // デバイス側データ
    std::array<cuda_unique_ptr<T[]>, NumVariables> device_data;
    std::array<cuda_unique_ptr<T[]>, NumVariables> device_grad;
    cuda_unique_ptr<T[]> device_result;
    
    // 結果格納用変数の数
    std::size_t result_size;
    
    TestBuffer(std::size_t result_count = 0) : result_size(result_count) {
        // ホストデータ初期化
        for (auto& data : host_data) {
            data.fill(T{});
        }
        for (auto& grad : host_grad) {
            grad.fill(T{});
        }
        
        if (result_size > 0) {
            host_result.resize(result_size, T{});
        }
        
        // デバイスメモリ確保
        for (std::size_t i = 0; i < NumVariables; ++i) {
            device_data[i] = makeCudaUniqueArray<T>(VarSize);
            device_grad[i] = makeCudaUniqueArray<T>(VarSize);
        }
        
        if (result_size > 0) {
            device_result = makeCudaUniqueArray<T>(result_size);
        }
    }
    
    // ホストからデバイスへデータ転送
    void toGpu() {
        for (std::size_t i = 0; i < NumVariables; ++i) {
            hipMemcpy(device_data[i].get(), host_data[i].data(), VarSize * sizeof(T), hipMemcpyHostToDevice);
            hipMemcpy(device_grad[i].get(), host_grad[i].data(), VarSize * sizeof(T), hipMemcpyHostToDevice);
        }
        hipDeviceSynchronize();
    }
    
    // デバイスからホストへ結果転送
    void toHost() {
        if (result_size > 0) {
            hipMemcpy(host_result.data(), device_result.get(), result_size * sizeof(T), hipMemcpyDeviceToHost);
        }
        
        // 勾配も取得（テスト検証用）
        for (std::size_t i = 0; i < NumVariables; ++i) {
            hipMemcpy(host_grad[i].data(), device_grad[i].get(), VarSize * sizeof(T), hipMemcpyDeviceToHost);
        }
        hipDeviceSynchronize();
    }
    
    // 指定したインデックスの変数データを設定
    void setData(std::size_t var_idx, std::size_t element_idx, T value) {
        if (var_idx < NumVariables && element_idx < VarSize) {
            host_data[var_idx][element_idx] = value;
        }
    }
    
    // 指定したインデックスの勾配データを設定
    void setGrad(std::size_t var_idx, std::size_t element_idx, T value) {
        if (var_idx < NumVariables && element_idx < VarSize) {
            host_grad[var_idx][element_idx] = value;
        }
    }
    
    // デバイスポインタ取得
    T* getDeviceData(std::size_t var_idx) {
        return var_idx < NumVariables ? device_data[var_idx].get() : nullptr;
    }
    
    T* getDeviceGrad(std::size_t var_idx) {
        return var_idx < NumVariables ? device_grad[var_idx].get() : nullptr;
    }
    
    T* getDeviceResult() {
        return device_result.get();
    }
    
    // 結果値取得
    T getResult(std::size_t idx) const {
        return idx < result_size ? host_result[idx] : T{};
    }
    
    // 勾配値取得
    T getGrad(std::size_t var_idx, std::size_t element_idx) const {
        return (var_idx < NumVariables && element_idx < VarSize) ? host_grad[var_idx][element_idx] : T{};
    }
};

// Operation テスト用のCUDAカーネル
template <typename T>
__global__ void test_operation_kernel(T* data1, T* grad1, T* data2, T* grad2, T* result) {
    // Variable作成
    Variable<T, 1> var1(data1, grad1);
    Variable<T, 1> var2(data2, grad2);
    
    // 値設定
    var1[0] = static_cast<T>(3.0);
    var2[0] = static_cast<T>(4.0);
    
    // add関数を使用してBinaryOperationを作成（内部でbufferを保持、自動でforward実行）
    auto op = op::add(var1, var2);
    
    // 結果を取得（forward計算は自動実行済み）
    result[0] = op.output()[0];
    
    // 出力に単位勾配を設定してbackward計算
    op.output().grad(0) = static_cast<T>(1.0);
    
    // backward計算
    op.backward();
    
    // 勾配結果を保存
    result[1] = var1.grad(0);  // dL/dvar1
    result[2] = var2.grad(0);  // dL/dvar2
}

class OperationTest : public ::testing::Test {
protected:
    void SetUp() override {
        hipError_t err = hipSetDevice(0);
        if (err != hipSuccess) {
            GTEST_SKIP() << "CUDA device not available";
        }
    }
};

TEST_F(OperationTest, BasicAddition) {
    using T = float;
    
    // テストバッファ作成（2変数、結果3個）
    TestBuffer<T, 2> buffer(3);
    
    // 勾配初期化（データは自動的にゼロ初期化される）
    buffer.toGpu();
    
    // カーネル実行
    test_operation_kernel<T><<<1, 1>>>(
        buffer.getDeviceData(0), buffer.getDeviceGrad(0),
        buffer.getDeviceData(1), buffer.getDeviceGrad(1),
        buffer.getDeviceResult());
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    
    // 結果をホストにコピー
    buffer.toHost();
    
    // 検証
    EXPECT_FLOAT_EQ(buffer.getResult(0), 7.0f);  // 3 + 4 = 7
    EXPECT_FLOAT_EQ(buffer.getResult(1), 1.0f);  // d(3+4)/d3 = 1
    EXPECT_FLOAT_EQ(buffer.getResult(2), 1.0f);  // d(3+4)/d4 = 1
}

// OperationRef テスト用のCUDAカーネル
template <typename T>
__global__ void test_operation_ref_kernel(T* data1, T* grad1, T* data2, T* grad2, T* output_data, T* output_grad, T* result) {
    // Variable作成
    Variable<T, 1> var1(data1, grad1);
    Variable<T, 1> var2(data2, grad2);
    Variable<T, 1> output_var(output_data, output_grad);
    
    // 値設定
    var1[0] = static_cast<T>(3.0);
    var2[0] = static_cast<T>(4.0);
    
    // add_ref関数を使用してBinaryOperationRefを作成（外部バッファを参照）
    auto op_ref = op::add_ref(var1, var2, output_var);
    
    // forward計算
    op_ref.forward();
    result[0] = op_ref.output()[0];
    
    // 出力に単位勾配を設定してbackward計算
    op_ref.output().grad(0) = static_cast<T>(1.0);
    
    // backward計算
    op_ref.backward();
    
    // 勾配結果を保存
    result[1] = var1.grad(0);  // dL/dvar1
    result[2] = var2.grad(0);  // dL/dvar2
}

TEST_F(OperationTest, BasicAdditionRef) {
    using T = float;
    
    // テストバッファ作成（3変数：入力2個+出力1個、結果3個）
    TestBuffer<T, 3> buffer(3);
    
    // データとバッファ初期化
    buffer.toGpu();
    
    // カーネル実行
    test_operation_ref_kernel<T><<<1, 1>>>(
        buffer.getDeviceData(0), buffer.getDeviceGrad(0),
        buffer.getDeviceData(1), buffer.getDeviceGrad(1),
        buffer.getDeviceData(2), buffer.getDeviceGrad(2),
        buffer.getDeviceResult());
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    
    // 結果をホストにコピー
    buffer.toHost();
    
    // 検証
    EXPECT_FLOAT_EQ(buffer.getResult(0), 7.0f);  // 3 + 4 = 7
    EXPECT_FLOAT_EQ(buffer.getResult(1), 1.0f);  // d(3+4)/d3 = 1
    EXPECT_FLOAT_EQ(buffer.getResult(2), 1.0f);  // d(3+4)/d4 = 1
}

TEST_F(OperationTest, ConceptCheck) {
    using namespace op;
    using Var1 = Variable<float, 1>;
    using AddLogicType = AddLogic<Var1, Var1>;
    using AddOp = BinaryOperation<AddLogicType::outputDim, AddLogicType, Var1, Var1>;
    using AddOpRef = BinaryOperationRef<AddLogicType::outputDim, AddLogicType, Var1, Var1>;
    
    // BinaryOperation型要件のチェック
    static_assert(std::is_same_v<AddOp::input1_type, Var1>);
    static_assert(std::is_same_v<AddOp::input2_type, Var1>);
    static_assert(std::is_same_v<AddOp::value_type, float>);
    static_assert(AddOp::output_size == 1);
    static_assert(AddLogicType::outputDim == 1);
    
    // BinaryOperationRef型要件のチェック
    static_assert(std::is_same_v<AddOpRef::input1_type, Var1>);
    static_assert(std::is_same_v<AddOpRef::input2_type, Var1>);
    static_assert(std::is_same_v<AddOpRef::value_type, float>);
    static_assert(AddOpRef::output_size == 1);
    
    // 出力型の一致確認
    static_assert(std::is_same_v<AddOp::output_type, AddOpRef::output_type>);
}

int main(int argc, char** argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}