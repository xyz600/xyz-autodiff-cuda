#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include "../include/variable.cuh"
#include "../include/util/cuda_unique_ptr.cuh"

class VariableOperatorTest : public ::testing::Test {
protected:
    void SetUp() override {
        // Check CUDA device availability
        int device_count;
        hipGetDeviceCount(&device_count);
        if (device_count == 0) {
            GTEST_SKIP() << "No CUDA devices available";
        }
    }
};

// Test buffer for Variable + constant operator
template <typename T, std::size_t N>
struct VariableOperatorTestBuffer {
    T input_data[N];
    T input_grad[N];
    T output_data[N];
    T constant_value;
    T output_grad[N];
};

// CUDA kernel to test Variable + constant operator
template <typename T, std::size_t N>
__global__ void test_variable_plus_constant_kernel(VariableOperatorTestBuffer<T, N>* buffer) {
    // Create VariableRef from buffer data and gradients
    xyz_autodiff::VariableRef<T, N> input(buffer->input_data, buffer->input_grad);
    
    // Zero gradient
    input.zero_grad();
    
    // Create operation and forward
    auto result_op = input + buffer->constant_value;
    result_op.forward();
    
    // Copy result to output data array
    for (std::size_t i = 0; i < N; ++i) {
        buffer->output_data[i] = result_op[i];
    }
    
    // Set gradient on output
    for (std::size_t i = 0; i < N; ++i) {
        result_op.add_grad(i, buffer->output_grad[i]);
    }
    
    // Backward pass
    result_op.backward();
}

TEST_F(VariableOperatorTest, VariablePlusConstantFloat) {
    using T = float;
    constexpr std::size_t N = 3;
    
    // Prepare host data
    VariableOperatorTestBuffer<T, N> host_buffer;
    
    // Initialize input values on host
    T input_values[N] = {1.0f, 2.0f, 3.0f};
    for (std::size_t i = 0; i < N; ++i) {
        host_buffer.input_data[i] = input_values[i];
    }
    
    // Set constant value
    host_buffer.constant_value = 2.5f;
    
    // Initialize input gradients to zero
    for (std::size_t i = 0; i < N; ++i) {
        host_buffer.input_grad[i] = 0.0f;
    }
    
    // Set output gradients (all 1.0 for simplicity)
    for (std::size_t i = 0; i < N; ++i) {
        host_buffer.output_grad[i] = 1.0f;
    }
    
    // Copy to device
    auto device_buffer = makeCudaUnique<VariableOperatorTestBuffer<T, N>>();
    hipMemcpy(device_buffer.get(), &host_buffer, sizeof(VariableOperatorTestBuffer<T, N>), hipMemcpyHostToDevice);
    
    // Launch kernel
    test_variable_plus_constant_kernel<T, N><<<1, 1>>>(device_buffer.get());
    hipDeviceSynchronize();
    
    // Copy result back
    hipMemcpy(&host_buffer, device_buffer.get(), sizeof(VariableOperatorTestBuffer<T, N>), hipMemcpyDeviceToHost);
    
    // Verify results on host
    T expected_output[N] = {1.0f + 2.5f, 2.0f + 2.5f, 3.0f + 2.5f};
    T expected_input_grad[N] = {1.0f, 1.0f, 1.0f}; // Since d/dx(x + c) = 1
    
    for (std::size_t i = 0; i < N; ++i) {
        EXPECT_NEAR(host_buffer.output_data[i], expected_output[i], 1e-6f) 
            << "Forward pass failed at index " << i;
        EXPECT_NEAR(host_buffer.input_grad[i], expected_input_grad[i], 1e-6f)
            << "Backward pass failed at index " << i;
    }
}

TEST_F(VariableOperatorTest, VariablePlusConstantDouble) {
    using T = double;
    constexpr std::size_t N = 4;
    
    // Prepare host data
    VariableOperatorTestBuffer<T, N> host_buffer;
    
    // Initialize input values on host
    T input_values[N] = {1.0, 2.0, 3.0, 4.0};
    for (std::size_t i = 0; i < N; ++i) {
        host_buffer.input_data[i] = input_values[i];
    }
    
    // Set constant value
    host_buffer.constant_value = -1.25;
    
    // Initialize input gradients to zero
    for (std::size_t i = 0; i < N; ++i) {
        host_buffer.input_grad[i] = 0.0;
    }
    
    // Set output gradients (all 1.0 for simplicity)
    for (std::size_t i = 0; i < N; ++i) {
        host_buffer.output_grad[i] = 1.0;
    }
    
    // Copy to device
    auto device_buffer = makeCudaUnique<VariableOperatorTestBuffer<T, N>>();
    hipMemcpy(device_buffer.get(), &host_buffer, sizeof(VariableOperatorTestBuffer<T, N>), hipMemcpyHostToDevice);
    
    // Launch kernel
    test_variable_plus_constant_kernel<T, N><<<1, 1>>>(device_buffer.get());
    hipDeviceSynchronize();
    
    // Copy result back
    hipMemcpy(&host_buffer, device_buffer.get(), sizeof(VariableOperatorTestBuffer<T, N>), hipMemcpyDeviceToHost);
    
    // Verify results on host
    T expected_output[N] = {1.0 + (-1.25), 2.0 + (-1.25), 3.0 + (-1.25), 4.0 + (-1.25)};
    T expected_input_grad[N] = {1.0, 1.0, 1.0, 1.0}; // Since d/dx(x + c) = 1
    
    for (std::size_t i = 0; i < N; ++i) {
        EXPECT_NEAR(host_buffer.output_data[i], expected_output[i], 1e-10) 
            << "Forward pass failed at index " << i;
        EXPECT_NEAR(host_buffer.input_grad[i], expected_input_grad[i], 1e-10)
            << "Backward pass failed at index " << i;
    }
}

// Test with zero constant
TEST_F(VariableOperatorTest, VariablePlusZeroConstant) {
    using T = float;
    constexpr std::size_t N = 2;
    
    // Prepare host data
    VariableOperatorTestBuffer<T, N> host_buffer;
    
    // Initialize input values on host
    T input_values[N] = {1.0f, 2.0f};
    for (std::size_t i = 0; i < N; ++i) {
        host_buffer.input_data[i] = input_values[i];
    }
    
    // Set constant value to zero
    host_buffer.constant_value = 0.0f;
    
    // Initialize input gradients to zero
    for (std::size_t i = 0; i < N; ++i) {
        host_buffer.input_grad[i] = 0.0f;
    }
    
    // Set output gradients (all 1.0 for simplicity)
    for (std::size_t i = 0; i < N; ++i) {
        host_buffer.output_grad[i] = 1.0f;
    }
    
    // Copy to device
    auto device_buffer = makeCudaUnique<VariableOperatorTestBuffer<T, N>>();
    hipMemcpy(device_buffer.get(), &host_buffer, sizeof(VariableOperatorTestBuffer<T, N>), hipMemcpyHostToDevice);
    
    // Launch kernel
    test_variable_plus_constant_kernel<T, N><<<1, 1>>>(device_buffer.get());
    hipDeviceSynchronize();
    
    // Copy result back
    hipMemcpy(&host_buffer, device_buffer.get(), sizeof(VariableOperatorTestBuffer<T, N>), hipMemcpyDeviceToHost);
    
    // Verify results on host (should be same as input since adding zero)
    T expected_output[N] = {1.0f, 2.0f};
    T expected_input_grad[N] = {1.0f, 1.0f}; // Since d/dx(x + c) = 1
    
    for (std::size_t i = 0; i < N; ++i) {
        EXPECT_NEAR(host_buffer.output_data[i], expected_output[i], 1e-6f) 
            << "Forward pass failed at index " << i;
        EXPECT_NEAR(host_buffer.input_grad[i], expected_input_grad[i], 1e-6f)
            << "Backward pass failed at index " << i;
    }
}

// Test buffer for specific size operations
struct VariableMinusTestBuffer {
    float input_data[3];
    float input_grad[3];
    float output_data[3];
    float constant_value;
    float output_grad[3];
};

// CUDA kernel to test Variable - constant operator
__global__ void test_variable_minus_constant_kernel(VariableMinusTestBuffer* buffer) {
    // Create VariableRef from buffer data and gradients
    xyz_autodiff::VariableRef<float, 3> input(buffer->input_data, buffer->input_grad);
    
    // Zero gradient
    input.zero_grad();
    
    // Create operation and forward
    auto result_op = input - buffer->constant_value;
    result_op.forward();
    
    // Copy result to output data array
    for (std::size_t i = 0; i < 3; ++i) {
        buffer->output_data[i] = result_op[i];
    }
    
    // Set gradient on output
    for (std::size_t i = 0; i < 3; ++i) {
        result_op.add_grad(i, buffer->output_grad[i]);
    }
    
    // Backward pass
    result_op.backward();
}

TEST_F(VariableOperatorTest, VariableMinusConstantFloat) {
    // Prepare host data
    VariableMinusTestBuffer host_buffer;
    
    // Initialize input values on host
    float input_values[3] = {5.0f, 4.0f, 3.0f};
    for (std::size_t i = 0; i < 3; ++i) {
        host_buffer.input_data[i] = input_values[i];
        host_buffer.input_grad[i] = 0.0f;
        host_buffer.output_grad[i] = 1.0f;
    }
    
    // Set constant value
    host_buffer.constant_value = 2.0f;
    
    // Copy to device
    auto device_buffer = makeCudaUnique<VariableMinusTestBuffer>();
    hipMemcpy(device_buffer.get(), &host_buffer, sizeof(VariableMinusTestBuffer), hipMemcpyHostToDevice);
    
    // Launch kernel
    test_variable_minus_constant_kernel<<<1, 1>>>(device_buffer.get());
    hipDeviceSynchronize();
    
    // Copy result back
    hipMemcpy(&host_buffer, device_buffer.get(), sizeof(VariableMinusTestBuffer), hipMemcpyDeviceToHost);
    
    // Verify results on host
    float expected_output[3] = {3.0f, 2.0f, 1.0f};
    float expected_input_grad[3] = {1.0f, 1.0f, 1.0f}; // Since d/dx(x - c) = 1
    
    for (std::size_t i = 0; i < 3; ++i) {
        EXPECT_NEAR(host_buffer.output_data[i], expected_output[i], 1e-6f) 
            << "Forward pass failed at index " << i;
        EXPECT_NEAR(host_buffer.input_grad[i], expected_input_grad[i], 1e-6f)
            << "Backward pass failed at index " << i;
    }
}

// Test buffer for multiply operation
struct VariableMultiplyTestBuffer {
    float input_data[2];
    float input_grad[2];
    float output_data[2];
    float constant_value;
    float output_grad[2];
};

// CUDA kernel to test Variable * constant operator
__global__ void test_variable_multiply_constant_kernel(VariableMultiplyTestBuffer* buffer) {
    // Create VariableRef from buffer data and gradients
    xyz_autodiff::VariableRef<float, 2> input(buffer->input_data, buffer->input_grad);
    
    // Zero gradient
    input.zero_grad();
    
    // Create operation and forward
    auto result_op = input * buffer->constant_value;
    result_op.forward();
    
    // Copy result to output data array
    for (std::size_t i = 0; i < 2; ++i) {
        buffer->output_data[i] = result_op[i];
    }
    
    // Set gradient on output
    for (std::size_t i = 0; i < 2; ++i) {
        result_op.add_grad(i, buffer->output_grad[i]);
    }
    
    // Backward pass
    result_op.backward();
}

TEST_F(VariableOperatorTest, VariableMultiplyConstantFloat) {
    // Prepare host data
    VariableMultiplyTestBuffer host_buffer;
    
    // Initialize input values on host
    float input_values[2] = {2.0f, 3.0f};
    for (std::size_t i = 0; i < 2; ++i) {
        host_buffer.input_data[i] = input_values[i];
        host_buffer.input_grad[i] = 0.0f;
        host_buffer.output_grad[i] = 1.0f;
    }
    
    // Set constant value
    host_buffer.constant_value = 1.5f;
    
    // Copy to device
    auto device_buffer = makeCudaUnique<VariableMultiplyTestBuffer>();
    hipMemcpy(device_buffer.get(), &host_buffer, sizeof(VariableMultiplyTestBuffer), hipMemcpyHostToDevice);
    
    // Launch kernel
    test_variable_multiply_constant_kernel<<<1, 1>>>(device_buffer.get());
    hipDeviceSynchronize();
    
    // Copy result back
    hipMemcpy(&host_buffer, device_buffer.get(), sizeof(VariableMultiplyTestBuffer), hipMemcpyDeviceToHost);
    
    // Verify results on host
    float expected_output[2] = {3.0f, 4.5f};
    float expected_input_grad[2] = {1.5f, 1.5f}; // Since d/dx(x * c) = c
    
    for (std::size_t i = 0; i < 2; ++i) {
        EXPECT_NEAR(host_buffer.output_data[i], expected_output[i], 1e-6f) 
            << "Forward pass failed at index " << i;
        EXPECT_NEAR(host_buffer.input_grad[i], expected_input_grad[i], 1e-6f)
            << "Backward pass failed at index " << i;
    }
}

// Test buffer for divide operation
struct VariableDivideTestBuffer {
    double input_data[2];
    double input_grad[2];
    double output_data[2];
    double constant_value;
    double output_grad[2];
};

// CUDA kernel to test Variable / constant operator
__global__ void test_variable_divide_constant_kernel(VariableDivideTestBuffer* buffer) {
    // Create VariableRef from buffer data and gradients
    xyz_autodiff::VariableRef<double, 2> input(buffer->input_data, buffer->input_grad);
    
    // Zero gradient
    input.zero_grad();
    
    // Create operation and forward
    auto result_op = input / buffer->constant_value;
    result_op.forward();
    
    // Copy result to output data array
    for (std::size_t i = 0; i < 2; ++i) {
        buffer->output_data[i] = result_op[i];
    }
    
    // Set gradient on output
    for (std::size_t i = 0; i < 2; ++i) {
        result_op.add_grad(i, buffer->output_grad[i]);
    }
    
    // Backward pass
    result_op.backward();
}

TEST_F(VariableOperatorTest, VariableDivideConstantDouble) {
    // Prepare host data
    VariableDivideTestBuffer host_buffer;
    
    // Initialize input values on host
    double input_values[2] = {6.0, 9.0};
    for (std::size_t i = 0; i < 2; ++i) {
        host_buffer.input_data[i] = input_values[i];
        host_buffer.input_grad[i] = 0.0;
        host_buffer.output_grad[i] = 1.0;
    }
    
    // Set constant value
    host_buffer.constant_value = 3.0;
    
    // Copy to device
    auto device_buffer = makeCudaUnique<VariableDivideTestBuffer>();
    hipMemcpy(device_buffer.get(), &host_buffer, sizeof(VariableDivideTestBuffer), hipMemcpyHostToDevice);
    
    // Launch kernel
    test_variable_divide_constant_kernel<<<1, 1>>>(device_buffer.get());
    hipDeviceSynchronize();
    
    // Copy result back
    hipMemcpy(&host_buffer, device_buffer.get(), sizeof(VariableDivideTestBuffer), hipMemcpyDeviceToHost);
    
    // Verify results on host
    double expected_output[2] = {2.0, 3.0};
    double expected_input_grad[2] = {1.0 / 3.0, 1.0 / 3.0}; // Since d/dx(x / c) = 1/c
    
    for (std::size_t i = 0; i < 2; ++i) {
        EXPECT_NEAR(host_buffer.output_data[i], expected_output[i], 1e-10) 
            << "Forward pass failed at index " << i;
        EXPECT_NEAR(host_buffer.input_grad[i], expected_input_grad[i], 1e-10)
            << "Backward pass failed at index " << i;
    }
}

