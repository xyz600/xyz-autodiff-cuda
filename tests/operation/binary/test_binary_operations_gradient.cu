#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "../../../include/variable.cuh"
#include "../../../include/concept/variable.cuh"
#include "../../../include/concept/operation_node.cuh"
#include "../../../include/operations/binary/add_logic.cuh"
#include "../../../include/operations/binary/sub_logic.cuh"
#include "../../../include/operations/binary/mul_logic.cuh"
#include "../../../include/operations/binary/div_logic.cuh"
#include "../../../include/util/cuda_unique_ptr.cuh"
#include "../../utility/binary_gradient_tester.cuh"

using namespace xyz_autodiff;

// ===========================================
// Static Assert Tests for Concept Compliance
// ===========================================

// Test types
using TestVector3 = Variable<3, float>;
using TestVectorRef3 = VariableRef<3, float>;
using TestVector2 = Variable<2, float>;
using TestVectorRef2 = VariableRef<2, float>;

// Binary operation types
using AddOp = BinaryOperation<3, op::AddLogic<TestVectorRef3, TestVectorRef3>, TestVectorRef3, TestVectorRef3>;
using SubOp = BinaryOperation<3, op::SubLogic<TestVectorRef3, TestVectorRef3>, TestVectorRef3, TestVectorRef3>;
using MulOp = BinaryOperation<3, op::MulLogic<TestVectorRef3, TestVectorRef3>, TestVectorRef3, TestVectorRef3>;
using DivOp = BinaryOperation<3, op::DivLogic<TestVectorRef3, TestVectorRef3>, TestVectorRef3, TestVectorRef3>;

// Static assertions for concept compliance
static_assert(VariableConcept<AddOp>, "AddOperation should satisfy VariableConcept");
static_assert(DifferentiableVariableConcept<AddOp>, "AddOperation should satisfy DifferentiableVariableConcept");
static_assert(OperationNode<AddOp>, "AddOperation should satisfy OperationNode");

static_assert(VariableConcept<SubOp>, "SubOperation should satisfy VariableConcept");
static_assert(DifferentiableVariableConcept<SubOp>, "SubOperation should satisfy DifferentiableVariableConcept");
static_assert(OperationNode<SubOp>, "SubOperation should satisfy OperationNode");

static_assert(VariableConcept<MulOp>, "MulOperation should satisfy VariableConcept");
static_assert(DifferentiableVariableConcept<MulOp>, "MulOperation should satisfy DifferentiableVariableConcept");
static_assert(OperationNode<MulOp>, "MulOperation should satisfy OperationNode");

static_assert(VariableConcept<DivOp>, "DivOperation should satisfy VariableConcept");
static_assert(DifferentiableVariableConcept<DivOp>, "DivOperation should satisfy DifferentiableVariableConcept");
static_assert(OperationNode<DivOp>, "DivOperation should satisfy OperationNode");

// Ensure Variable is NOT an OperationNode
static_assert(!OperationNode<TestVector3>, "Variable should NOT be OperationNode");

// ===========================================
// Test Class
// ===========================================

class BinaryOperationsGradientTest : public ::testing::Test {
protected:
    void SetUp() override {
        int device_count;
        hipGetDeviceCount(&device_count);
        if (device_count == 0) {
            GTEST_SKIP() << "No CUDA devices available";
        }
    }
};

// ===========================================
// Forward Pass Tests
// ===========================================

__global__ void test_add_forward_kernel(float* result) {
    float input1_data[3] = {2.0f, 3.0f, 1.5f};
    float input1_grad[3] = {0.0f, 0.0f, 0.0f};
    float input2_data[3] = {1.0f, -1.0f, 2.5f};
    float input2_grad[3] = {0.0f, 0.0f, 0.0f};
    
    VariableRef<3, float> input1(input1_data, input1_grad);
    VariableRef<3, float> input2(input2_data, input2_grad);
    
    auto add_op = op::add(input1, input2);
    add_op.forward();
    
    // Expected: [3.0, 2.0, 4.0]
    float tolerance = 1e-6f;
    bool success = (fabsf(add_op[0] - 3.0f) < tolerance &&
                   fabsf(add_op[1] - 2.0f) < tolerance &&
                   fabsf(add_op[2] - 4.0f) < tolerance);
    
    result[0] = success ? 1.0f : 0.0f;
}

__global__ void test_sub_forward_kernel(float* result) {
    float input1_data[3] = {5.0f, 3.0f, 1.0f};
    float input1_grad[3] = {0.0f, 0.0f, 0.0f};
    float input2_data[3] = {2.0f, 1.0f, 0.5f};
    float input2_grad[3] = {0.0f, 0.0f, 0.0f};
    
    VariableRef<3, float> input1(input1_data, input1_grad);
    VariableRef<3, float> input2(input2_data, input2_grad);
    
    auto sub_op = op::sub(input1, input2);
    sub_op.forward();
    
    // Expected: [3.0, 2.0, 0.5]
    float tolerance = 1e-6f;
    bool success = (fabsf(sub_op[0] - 3.0f) < tolerance &&
                   fabsf(sub_op[1] - 2.0f) < tolerance &&
                   fabsf(sub_op[2] - 0.5f) < tolerance);
    
    result[0] = success ? 1.0f : 0.0f;
}

__global__ void test_mul_forward_kernel(float* result) {
    float input1_data[3] = {2.0f, 3.0f, -1.0f};
    float input1_grad[3] = {0.0f, 0.0f, 0.0f};
    float input2_data[3] = {1.5f, -2.0f, 4.0f};
    float input2_grad[3] = {0.0f, 0.0f, 0.0f};
    
    VariableRef<3, float> input1(input1_data, input1_grad);
    VariableRef<3, float> input2(input2_data, input2_grad);
    
    auto mul_op = op::mul(input1, input2);
    mul_op.forward();
    
    // Expected: [3.0, -6.0, -4.0]
    float tolerance = 1e-6f;
    bool success = (fabsf(mul_op[0] - 3.0f) < tolerance &&
                   fabsf(mul_op[1] - (-6.0f)) < tolerance &&
                   fabsf(mul_op[2] - (-4.0f)) < tolerance);
    
    result[0] = success ? 1.0f : 0.0f;
}

__global__ void test_div_forward_kernel(float* result) {
    float input1_data[3] = {6.0f, -8.0f, 10.0f};
    float input1_grad[3] = {0.0f, 0.0f, 0.0f};
    float input2_data[3] = {2.0f, -4.0f, 5.0f};
    float input2_grad[3] = {0.0f, 0.0f, 0.0f};
    
    VariableRef<3, float> input1(input1_data, input1_grad);
    VariableRef<3, float> input2(input2_data, input2_grad);
    
    auto div_op = op::div(input1, input2);
    div_op.forward();
    
    // Expected: [3.0, 2.0, 2.0]
    float tolerance = 1e-6f;
    bool success = (fabsf(div_op[0] - 3.0f) < tolerance &&
                   fabsf(div_op[1] - 2.0f) < tolerance &&
                   fabsf(div_op[2] - 2.0f) < tolerance);
    
    result[0] = success ? 1.0f : 0.0f;
}

// ===========================================
// Test Cases
// ===========================================

TEST_F(BinaryOperationsGradientTest, AddForwardPass) {
    auto device_result = makeCudaUnique<float>();
    test_add_forward_kernel<<<1, 1>>>(device_result.get());
    hipDeviceSynchronize();
    
    float host_result;
    hipMemcpy(&host_result, device_result.get(), sizeof(float), hipMemcpyDeviceToHost);
    EXPECT_EQ(host_result, 1.0f);
}

TEST_F(BinaryOperationsGradientTest, SubForwardPass) {
    auto device_result = makeCudaUnique<float>();
    test_sub_forward_kernel<<<1, 1>>>(device_result.get());
    hipDeviceSynchronize();
    
    float host_result;
    hipMemcpy(&host_result, device_result.get(), sizeof(float), hipMemcpyDeviceToHost);
    EXPECT_EQ(host_result, 1.0f);
}

TEST_F(BinaryOperationsGradientTest, MulForwardPass) {
    auto device_result = makeCudaUnique<float>();
    test_mul_forward_kernel<<<1, 1>>>(device_result.get());
    hipDeviceSynchronize();
    
    float host_result;
    hipMemcpy(&host_result, device_result.get(), sizeof(float), hipMemcpyDeviceToHost);
    EXPECT_EQ(host_result, 1.0f);
}

TEST_F(BinaryOperationsGradientTest, DivForwardPass) {
    auto device_result = makeCudaUnique<float>();
    test_div_forward_kernel<<<1, 1>>>(device_result.get());
    hipDeviceSynchronize();
    
    float host_result;
    hipMemcpy(&host_result, device_result.get(), sizeof(float), hipMemcpyDeviceToHost);
    EXPECT_EQ(host_result, 1.0f);
}

// ===========================================
// Gradient Verification Tests
// ===========================================

TEST_F(BinaryOperationsGradientTest, AddGradientVerification) {
    using Logic = op::AddLogic<VariableRef<3, double>, VariableRef<3, double>>;
    test::BinaryGradientTester<Logic, 3, 3, 3>::test_custom(
        "AddLogic", 
        50,      // num_tests
        1e-5,    // tolerance
        1e-7,    // delta
        -5.0,    // input_min
        5.0      // input_max
    );
}

TEST_F(BinaryOperationsGradientTest, SubGradientVerification) {
    using Logic = op::SubLogic<VariableRef<3, double>, VariableRef<3, double>>;
    test::BinaryGradientTester<Logic, 3, 3, 3>::test_custom(
        "SubLogic", 
        50,      // num_tests
        1e-5,    // tolerance
        1e-7,    // delta
        -5.0,    // input_min
        5.0      // input_max
    );
}

TEST_F(BinaryOperationsGradientTest, MulGradientVerification) {
    using Logic = op::MulLogic<VariableRef<3, double>, VariableRef<3, double>>;
    test::BinaryGradientTester<Logic, 3, 3, 3>::test_custom(
        "MulLogic", 
        50,      // num_tests
        1e-5,    // tolerance
        1e-7,    // delta
        -3.0,    // input_min (smaller range to avoid overflow)
        3.0      // input_max
    );
}

TEST_F(BinaryOperationsGradientTest, DivGradientVerification) {
    using Logic = op::DivLogic<VariableRef<3, double>, VariableRef<3, double>>;
    test::BinaryGradientTester<Logic, 3, 3, 3>::test_custom(
        "DivLogic", 
        50,      // num_tests
        1e-5,    // tolerance (minimum allowed for double precision)
        1e-6,    // delta
        0.1,     // input_min (avoid division by zero)
        5.0      // input_max
    );
}

// Test with different dimensions
TEST_F(BinaryOperationsGradientTest, AddGradientVerification2D) {
    using Logic = op::AddLogic<VariableRef<2, double>, VariableRef<2, double>>;
    test::BinaryGradientTester<Logic, 2, 2, 2>::test_custom(
        "AddLogic2D", 
        30,      // num_tests
        1e-5,    // tolerance
        1e-7,    // delta
        -10.0,   // input_min
        10.0     // input_max
    );
}

TEST_F(BinaryOperationsGradientTest, MulGradientVerification1D) {
    using Logic = op::MulLogic<VariableRef<1, double>, VariableRef<1, double>>;
    test::BinaryGradientTester<Logic, 1, 1, 1>::test_custom(
        "MulLogic1D", 
        30,      // num_tests
        1e-5,    // tolerance
        1e-7,    // delta
        -2.0,    // input_min
        2.0      // input_max
    );
}