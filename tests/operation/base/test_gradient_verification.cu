#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include "../../utility/gradient_test_utility.cuh"
#include "../../../include/operations/unary/sigmoid_logic.cuh"
#include "../../../include/operations/unary/exp_logic.cuh"
#include "../../../include/operations/binary/add_logic.cuh"
#include "../../../include/operations/binary/mul_logic.cuh"

using namespace xyz_autodiff;
using namespace xyz_autodiff::test;

// SigmoidLogicのテスト (UnaryOperation)
TEST_UNARY_GRADIENT(SigmoidLogic<3>, 3, 3, SigmoidLogic3D)
TEST_UNARY_GRADIENT(SigmoidLogic<1>, 1, 1, SigmoidLogic1D)
TEST_UNARY_GRADIENT(SigmoidLogic<5>, 5, 5, SigmoidLogic5D)

// ExpLogicのテスト (UnaryOperation)
TEST_UNARY_GRADIENT(ExpLogic<3>, 3, 3, ExpLogic3D)
TEST_UNARY_GRADIENT(ExpLogic<1>, 1, 1, ExpLogic1D)
TEST_UNARY_GRADIENT(ExpLogic<5>, 5, 5, ExpLogic5D)

// AddLogicのテスト (BinaryOperation) - 一旦コメントアウト（型推論の問題を解決するため）
// TEST_BINARY_GRADIENT(xyz_autodiff::op::AddLogic<VariableRef<double, 1>, VariableRef<double, 1>>, 1, 1, 1, AddLogic1D)

// より大きな次元でのテスト
TEST_UNARY_GRADIENT(SigmoidLogic<10>, 10, 10, SigmoidLogic10D)
TEST_UNARY_GRADIENT(ExpLogic<10>, 10, 10, ExpLogic10D)

// 特殊ケース: 異なる入力・出力次元
TEST_UNARY_GRADIENT(SigmoidLogic<7>, 7, 7, SigmoidLogic7D)
TEST_UNARY_GRADIENT(ExpLogic<8>, 8, 8, ExpLogic8D)

// より複雑なBinaryOperationテスト（将来的にAddLogic以外を追加する際に使用）
// TEST_BINARY_GRADIENT(xyz_autodiff::op::AddLogic<VariableRef<double, 3>, VariableRef<double, 3>>, 3, 3, 1, AddLogic3D)

// スケーラビリティテスト
TEST_UNARY_GRADIENT(SigmoidLogic<20>, 20, 20, SigmoidLogicLarge)
TEST_UNARY_GRADIENT(ExpLogic<20>, 20, 20, ExpLogicLarge)

class GradientVerificationTest : public ::testing::Test {
protected:
    void SetUp() override {
        hipError_t err = hipSetDevice(0);
        if (err != hipSuccess) {
            GTEST_SKIP() << "CUDA device not available";
        }
    }
};

// 手動テスト例: より詳細な診断が必要な場合
TEST_F(GradientVerificationTest, ManualSigmoidTest) {
    // より詳細なテストロジックがここに書ける
    // 例: 特定の入力値での詳細なデバッグ
    xyz_autodiff::test::UnaryGradientTester<SigmoidLogic<2>, 2, 2>::test("ManualSigmoidTest");
}

TEST_F(GradientVerificationTest, ManualExpTest) {
    xyz_autodiff::test::UnaryGradientTester<ExpLogic<2>, 2, 2>::test("ManualExpTest");
}

// エッジケーステスト
TEST_F(GradientVerificationTest, EdgeCaseSingleDimension) {
    xyz_autodiff::test::UnaryGradientTester<SigmoidLogic<1>, 1, 1>::test("EdgeCaseSingleDimension");
}

// BinaryOperationのテスト（element-wise operations）
// Type aliases to handle commas in template arguments
using MulLogic1D_t = xyz_autodiff::op::MulLogic<VariableRef<double, 1>, VariableRef<double, 1>>;
using AddLogic1D_t = xyz_autodiff::op::AddLogic<VariableRef<double, 1>, VariableRef<double, 1>>;
using MulLogic3D_t = xyz_autodiff::op::MulLogic<VariableRef<double, 3>, VariableRef<double, 3>>;
using AddLogic3D_t = xyz_autodiff::op::AddLogic<VariableRef<double, 3>, VariableRef<double, 3>>;
using MulLogic5D_t = xyz_autodiff::op::MulLogic<VariableRef<double, 5>, VariableRef<double, 5>>;
using AddLogic5D_t = xyz_autodiff::op::AddLogic<VariableRef<double, 5>, VariableRef<double, 5>>;

// 1次元
TEST_BINARY_GRADIENT(MulLogic1D_t, 1, 1, 1, MulLogic1D)
TEST_BINARY_GRADIENT(AddLogic1D_t, 1, 1, 1, AddLogic1D)

// 3次元
TEST_BINARY_GRADIENT(MulLogic3D_t, 3, 3, 3, MulLogic3D)
TEST_BINARY_GRADIENT(AddLogic3D_t, 3, 3, 3, AddLogic3D)

// 5次元
TEST_BINARY_GRADIENT(MulLogic5D_t, 5, 5, 5, MulLogic5D)
TEST_BINARY_GRADIENT(AddLogic5D_t, 5, 5, 5, AddLogic5D)

int main(int argc, char** argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}