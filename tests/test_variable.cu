#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "../include/variable.cuh"
// #include "../include/concept/variable.cuh"  // CUDA compiler concept limitations

using namespace xyz_autodiff;

// Variable テスト用のCUDAカーネル
template <typename T, std::size_t N>
__global__ void test_variable_kernel(T* data, T* grad, T* output) {
    // Variable作成
    Variable<T, N> var(data, grad);
    
    // データアクセステスト
    for (std::size_t i = 0; i < N; ++i) {
        var[i] = static_cast<T>(i + 1);  // 1, 2, 3, ...
    }
    
    // 勾配テスト
    for (std::size_t i = 0; i < N; ++i) {
        var.grad(i) = static_cast<T>(i * 2);  // 0, 2, 4, ...
    }
    
    // アクセサテスト
    T* data_ptr = var.data();
    T* grad_ptr = var.grad();
    
    // 結果をoutputに保存（検証用）
    for (std::size_t i = 0; i < N; ++i) {
        output[i] = data_ptr[i];           // データ値
        output[N + i] = grad_ptr[i];       // 勾配値
    }
}

template <typename T, std::size_t N>
__global__ void test_variable_operations_kernel(T* data, T* grad, T* grad_values, T* output) {
    Variable<T, N> var(data, grad);
    
    // zero_gradテスト
    var.zero_grad();
    
    // accumulate_gradテスト
    var.accumulate_grad(grad_values);
    
    // 結果を保存
    for (std::size_t i = 0; i < N; ++i) {
        output[i] = var.grad(i);
    }
}

class VariableTest : public ::testing::Test {
protected:
    void SetUp() override {
        // CUDA初期化チェック
        hipError_t err = hipSetDevice(0);
        if (err != hipSuccess) {
            GTEST_SKIP() << "CUDA device not available";
        }
    }
};

TEST_F(VariableTest, BasicConstruction) {
    constexpr std::size_t N = 4;
    using T = float;
    
    // ホストメモリ
    std::vector<T> host_data(N, 0);
    std::vector<T> host_grad(N, 0);
    std::vector<T> host_output(2 * N, 0);
    
    // デバイスメモリ確保
    T* device_data;
    T* device_grad;
    T* device_output;
    
    ASSERT_EQ(hipMalloc(&device_data, N * sizeof(T)), hipSuccess);
    ASSERT_EQ(hipMalloc(&device_grad, N * sizeof(T)), hipSuccess);
    ASSERT_EQ(hipMalloc(&device_output, 2 * N * sizeof(T)), hipSuccess);
    
    // カーネル実行
    test_variable_kernel<T, N><<<1, 1>>>(device_data, device_grad, device_output);
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    
    // 結果をホストにコピー
    ASSERT_EQ(hipMemcpy(host_output.data(), device_output, 2 * N * sizeof(T), hipMemcpyDeviceToHost), hipSuccess);
    
    // 結果検証
    for (std::size_t i = 0; i < N; ++i) {
        EXPECT_FLOAT_EQ(host_output[i], static_cast<T>(i + 1));        // データ値
        EXPECT_FLOAT_EQ(host_output[N + i], static_cast<T>(i * 2));    // 勾配値
    }
    
    // メモリ解放
    hipFree(device_data);
    hipFree(device_grad);
    hipFree(device_output);
}

TEST_F(VariableTest, GradientOperations) {
    constexpr std::size_t N = 3;
    using T = float;
    
    // ホストメモリ
    std::vector<T> host_data(N, 0);
    std::vector<T> host_grad(N, 1.0f);  // 初期値1.0
    std::vector<T> host_grad_values = {2.0f, 3.0f, 4.0f};
    std::vector<T> host_output(N, 0);
    
    // デバイスメモリ確保
    T* device_data;
    T* device_grad;
    T* device_grad_values;
    T* device_output;
    
    ASSERT_EQ(hipMalloc(&device_data, N * sizeof(T)), hipSuccess);
    ASSERT_EQ(hipMalloc(&device_grad, N * sizeof(T)), hipSuccess);
    ASSERT_EQ(hipMalloc(&device_grad_values, N * sizeof(T)), hipSuccess);
    ASSERT_EQ(hipMalloc(&device_output, N * sizeof(T)), hipSuccess);
    
    // データをデバイスにコピー
    ASSERT_EQ(hipMemcpy(device_grad, host_grad.data(), N * sizeof(T), hipMemcpyHostToDevice), hipSuccess);
    ASSERT_EQ(hipMemcpy(device_grad_values, host_grad_values.data(), N * sizeof(T), hipMemcpyHostToDevice), hipSuccess);
    
    // カーネル実行
    test_variable_operations_kernel<T, N><<<1, 1>>>(device_data, device_grad, device_grad_values, device_output);
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    
    // 結果をホストにコピー
    ASSERT_EQ(hipMemcpy(host_output.data(), device_output, N * sizeof(T), hipMemcpyDeviceToHost), hipSuccess);
    
    // zero_grad → accumulate_gradの結果を検証
    for (std::size_t i = 0; i < N; ++i) {
        EXPECT_FLOAT_EQ(host_output[i], host_grad_values[i]);
    }
    
    // メモリ解放
    hipFree(device_data);
    hipFree(device_grad);
    hipFree(device_grad_values);
    hipFree(device_output);
}

TEST_F(VariableTest, ConceptCheck) {
    // コンパイル時概念チェック (CUDA compiler limitations)
    // static_assert(concept::Variable<Variable<float, 4>>);
    // static_assert(concept::DifferentiableVariable<Variable<float, 4>>);
    
    // サイズチェック
    EXPECT_EQ((xyz_autodiff::Variable<float, 4>::size), 4);
    EXPECT_EQ((xyz_autodiff::Variable<double, 10>::size), 10);
}

int main(int argc, char** argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}