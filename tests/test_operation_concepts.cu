#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#include "../include/variable.cuh"
#include "../include/operations/binary/add_logic.cuh"
#include "../include/operations/binary/mul_logic.cuh"
#include "../include/operations/unary/exp_logic.cuh"
#include "../include/operations/unary/quaternion_to_rotation_matrix_logic.cuh"
#include "../include/concept/variable.cuh"
#include "../include/concept/operation_node.cuh"
#include "../include/util/cuda_unique_ptr.cuh"

using namespace xyz_autodiff;

// テスト用のOperation型を定義  
using TestVariable = VariableRef<float, 2>;
using TestUnaryOp = UnaryOperation<2, ExpLogic<2>, TestVariable>;
using TestBinaryOp = BinaryOperation<2, op::MulLogic<TestVariable, TestVariable>, TestVariable, TestVariable>;

// QuaternionToRotationMatrix用のテスト型
using TestQuaternion = Variable<float, 4>;
using TestQuatToMatOp = UnaryOperation<9, QuaternionToRotationMatrixLogic<4>, TestQuaternion>;

class OperationConceptTest : public ::testing::Test {
protected:
    void SetUp() override {
        hipError_t err = hipSetDevice(0);
        if (err != hipSuccess) {
            GTEST_SKIP() << "CUDA device not available";
        }
    }
};

// Variable関連のstatic_assert
static_assert(VariableConcept<Variable<float, 3>>, 
    "Variable should satisfy VariableConcept");
static_assert(VariableConcept<VariableRef<double, 5>>, 
    "VariableRef should satisfy VariableConcept");

static_assert(DifferentiableVariableConcept<Variable<float, 3>>, 
    "Variable should satisfy DifferentiableVariableConcept");
static_assert(DifferentiableVariableConcept<VariableRef<double, 5>>, 
    "VariableRef should satisfy DifferentiableVariableConcept");

// Operation関連のstatic_assert
static_assert(VariableConcept<TestUnaryOp>, 
    "UnaryOperation should satisfy VariableConcept");
static_assert(VariableConcept<TestBinaryOp>, 
    "BinaryOperation should satisfy VariableConcept");

static_assert(DifferentiableVariableConcept<TestUnaryOp>, 
    "UnaryOperation should satisfy DifferentiableVariableConcept");
static_assert(DifferentiableVariableConcept<TestBinaryOp>, 
    "BinaryOperation should satisfy DifferentiableVariableConcept");

static_assert(OperationNode<TestUnaryOp>, 
    "UnaryOperation should satisfy OperationNode");
static_assert(OperationNode<TestBinaryOp>, 
    "BinaryOperation should satisfy OperationNode");

// Variable は OperationNode ではないことを確認
static_assert(!OperationNode<Variable<float, 3>>, 
    "Variable should NOT satisfy OperationNode");
static_assert(!OperationNode<VariableRef<double, 5>>, 
    "VariableRef should NOT satisfy OperationNode");

// 具体的な型でのテスト
using TestVariable4d = Variable<double, 4>;
using TestVariable2f = Variable<float, 2>;
static_assert(VariableConcept<UnaryOperation<4, ExpLogic<4>, TestVariable4d>>, 
    "Specific UnaryOperation should satisfy VariableConcept");
static_assert(VariableConcept<BinaryOperation<2, op::MulLogic<TestVariable2f, TestVariable2f>, TestVariable2f, TestVariable2f>>, 
    "Specific BinaryOperation should satisfy VariableConcept");

// QuaternionToRotationMatrix Operation関連のstatic_assert
static_assert(VariableConcept<TestQuatToMatOp>, 
    "QuaternionToRotationMatrix Operation should satisfy VariableConcept");
static_assert(DifferentiableVariableConcept<TestQuatToMatOp>, 
    "QuaternionToRotationMatrix Operation should satisfy DifferentiableVariableConcept");
static_assert(OperationNode<TestQuatToMatOp>, 
    "QuaternionToRotationMatrix Operation should satisfy OperationNode");

// 具体的なQuaternionToRotationMatrix型でのテスト
using TestQuaternionFloat = Variable<float, 4>;
using TestQuaternionDouble = Variable<double, 4>;
static_assert(VariableConcept<UnaryOperation<9, QuaternionToRotationMatrixLogic<4>, TestQuaternionFloat>>, 
    "Float QuaternionToRotationMatrix Operation should satisfy VariableConcept");
static_assert(VariableConcept<UnaryOperation<9, QuaternionToRotationMatrixLogic<4>, TestQuaternionDouble>>, 
    "Double QuaternionToRotationMatrix Operation should satisfy VariableConcept");
static_assert(OperationNode<UnaryOperation<9, QuaternionToRotationMatrixLogic<4>, TestQuaternionFloat>>, 
    "Float QuaternionToRotationMatrix Operation should satisfy OperationNode");
static_assert(OperationNode<UnaryOperation<9, QuaternionToRotationMatrixLogic<4>, TestQuaternionDouble>>, 
    "Double QuaternionToRotationMatrix Operation should satisfy OperationNode");

// QuaternionToRotationMatrixLogic自体のテスト（型チェック）
static_assert(QuaternionToRotationMatrixLogic<4>::outputDim == 9, 
    "QuaternionToRotationMatrixLogic should have outputDim = 9");

// Quaternion変数は OperationNode ではないことを再確認
static_assert(!OperationNode<TestQuaternionFloat>, 
    "Quaternion Variable should NOT satisfy OperationNode");
static_assert(!OperationNode<TestQuaternionDouble>, 
    "Quaternion Variable should NOT satisfy OperationNode");

TEST_F(OperationConceptTest, ConceptComplianceBasicTest) {
    // このテストは主にコンパイル時の確認のためのもの
    // static_assertがすべて通ればテストは成功
    EXPECT_TRUE(true);
}

// Variableのzero_gradテスト用カーネル
template<typename T>
__global__ void test_variable_zero_grad_kernel(T* result) {
    Variable<float, 3> var;
    var.zero_grad(); // コンパイルが通ることを確認
    
    VariableRef<float, 3> var_ref(var.data(), var.grad());
    var_ref.zero_grad(); // コンパイルが通ることを確認
    
    *result = 1.0f; // 成功マーカー
}

TEST_F(OperationConceptTest, VariableZeroGradInterface) {
    auto device_result = makeCudaUnique<float>();
    
    test_variable_zero_grad_kernel<<<1, 1>>>(device_result.get());
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    
    float host_result;
    ASSERT_EQ(hipMemcpy(&host_result, device_result.get(), sizeof(float), hipMemcpyDeviceToHost), hipSuccess);
    EXPECT_EQ(host_result, 1.0f);
}

// Operationのインターフェーステスト用カーネル
__global__ void test_operation_interface_kernel(float* result) {
    float data1[2] = {1.0f, 2.0f};
    float grad1[2] = {0.0f, 0.0f};
    float data2[2] = {3.0f, 4.0f};
    float grad2[2] = {0.0f, 0.0f};
    
    VariableRef<float, 2> var1(data1, grad1);
    VariableRef<float, 2> var2(data2, grad2);
    
    // UnaryOperationを作成（ExpLogicを使用）
    ExpLogic<2> logic;
    UnaryOperation<2, ExpLogic<2>, VariableRef<float, 2>> op(logic, var1);
    
    // VariableConceptのインターフェースが使えることを確認
    op.zero_grad();  // zero_grad
    constexpr auto size = op.size;  // size
    auto* data = op.data();  // data()
    auto* grad = op.grad();  // grad()
    auto value = op[0];  // operator[]
    auto grad_value = op.grad(0);  // grad(size_t)
    
    // 結果を設定（サイズが正しいことを確認）
    *result = (size == 2 && data != nullptr && grad != nullptr) ? 1.0f : 0.0f;
}

TEST_F(OperationConceptTest, OperationVariableInterface) {
    auto device_result = makeCudaUnique<float>();
    
    test_operation_interface_kernel<<<1, 1>>>(device_result.get());
    ASSERT_EQ(hipDeviceSynchronize(), hipSuccess);
    
    float host_result;
    ASSERT_EQ(hipMemcpy(&host_result, device_result.get(), sizeof(float), hipMemcpyDeviceToHost), hipSuccess);
    EXPECT_EQ(host_result, 1.0f);
}

int main(int argc, char** argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}