#include "xyz_autodiff/tensor.h"
#include <hip/hip_runtime.h>

namespace xyz_autodiff {

Tensor::Tensor(int size) : size_(size) {
    hipMalloc(&data_, size * sizeof(float));
}

Tensor::~Tensor() {
    if (data_) {
        hipFree(data_);
    }
}

}