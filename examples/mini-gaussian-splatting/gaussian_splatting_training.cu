#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <sstream>
#include <random>
#include <chrono>

#include "image_utils.h"
#include "gaussian_parameters.h"
#include "gaussian_splatting_kernel.cuh"
#include "../../include/util/cuda_unique_ptr.cuh"

//using namespace xyz_autodiff;

class GaussianSplattingTrainer {
private:
    // Image data
    ImageData target_image;
    cuda_unique_ptr<float[]> device_target_image;
    cuda_unique_ptr<PixelOutput[]> device_output_image;
    cuda_unique_ptr<float> device_total_loss;
    
    // Gaussian collection
    GaussianCollection gaussians;
    
    // Training parameters
    float learning_rate;
    int max_iterations;
    int save_interval;
    
    // Random number generator
    std::mt19937 rng;
    
public:
    GaussianSplattingTrainer(float lr = 0.01f, int max_iter = 1000, int save_freq = 50)
        : learning_rate(lr), max_iterations(max_iter), save_interval(save_freq)
    {
        // Initialize random seed
        rng.seed(std::chrono::steady_clock::now().time_since_epoch().count());
    }
    
    ~GaussianSplattingTrainer() {
        // cuda_unique_ptr handles automatic cleanup
    }
    
    bool load_target_image(const std::string& filename) {
        std::cout << "Loading target image: " << filename << std::endl;
        
        if (!load_image(filename, target_image)) {
            std::cout << "Failed to load image, creating test image instead..." << std::endl;
            target_image = create_test_image(256, 256);
        }
        
        std::cout << "Target image: " << target_image.width << "x" << target_image.height 
                  << " (" << target_image.channels << " channels)" << std::endl;
        
        // Allocate device memory for target image using CUDA unique pointers
        int image_size = target_image.width * target_image.height * target_image.channels;
        device_target_image = makeCudaUniqueArray<float>(image_size);
        
        // Copy target image to device
        hipError_t err = hipMemcpy(device_target_image.get(), target_image.data.data(),
                                    image_size * sizeof(float), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "Failed to copy target image to device: " << hipGetErrorString(err) << std::endl;
            return false;
        }
        
        // Allocate device memory for output image
        int pixel_count = target_image.width * target_image.height;
        device_output_image = makeCudaUniqueArray<PixelOutput>(pixel_count);
        
        // Allocate device memory for total loss accumulator
        device_total_loss = makeCudaUnique<float>();
        
        return true;
    }
    
    void initialize_gaussians() {
        std::cout << "Initializing Gaussians..." << std::endl;
        gaussians.initialize_random(target_image.width, target_image.height, rng);
        gaussians.upload_to_device();
        std::cout << "Gaussian initialization complete." << std::endl;
    }
    
    void save_current_rendering(int iteration) {
        // Download output image from device
        int pixel_count = target_image.width * target_image.height;
        std::vector<PixelOutput> host_output(pixel_count);
        
        hipError_t err = hipMemcpy(host_output.data(), device_output_image.get(),
                                     pixel_count * sizeof(PixelOutput), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            std::cerr << "Failed to download output image: " << hipGetErrorString(err) << std::endl;
            return;
        }
        
        // Convert to ImageData format
        ImageData output_image(target_image.width, target_image.height, 3);
        for (int i = 0; i < pixel_count; i++) {
            output_image.data[i * 3 + 0] = host_output[i].color[0];
            output_image.data[i * 3 + 1] = host_output[i].color[1];
            output_image.data[i * 3 + 2] = host_output[i].color[2];
        }
        
        // Save as JPEG
        std::ostringstream filename;
        filename << "output/iteration_" << std::setfill('0') << std::setw(6) << iteration << ".jpg";
        save_image_jpeg(filename.str(), output_image);
    }
    
    void train() {
        std::cout << "\\n=== Starting Gaussian Splatting Training ===" << std::endl;
        std::cout << "Target image: " << target_image.width << "x" << target_image.height << std::endl;
        std::cout << "Gaussians: " << GaussianCollection::NUM_GAUSSIANS << std::endl;
        std::cout << "Learning rate: " << learning_rate << std::endl;
        std::cout << "Max iterations: " << max_iterations << std::endl;
        
        // Create output directory
        const auto status = system("mkdir -p output");
        if (status != 0) {
            std::cerr << "[warning]: failed to make directory";
        }
        
        // Save initial target image
        save_image_jpeg("output/target.jpg", target_image);
        
        for (int iteration = 0; iteration < 1000; iteration++) {
            auto start_time = std::chrono::high_resolution_clock::now();
            
            // Clear gradients and reset total loss
            gaussians.zero_gradients();
            gaussians.upload_to_device();
            
            // Zero the total loss accumulator
            float zero_loss = 0.0f;
            hipMemcpy(device_total_loss.get(), &zero_loss, sizeof(float), hipMemcpyHostToDevice);
            
            // Run forward and backward pass with atomic L1-norm accumulation
            launch_gaussian_splatting(
                gaussians.device_params.get(),
                gaussians.device_grads.get(),
                device_target_image.get(),
                device_output_image.get(),
                device_total_loss.get(),
                target_image.width,
                target_image.height,
                GaussianCollection::NUM_GAUSSIANS
            );

            // Download total loss from device
            float total_loss = 0.0f;
            hipMemcpy(&total_loss, device_total_loss.get(), sizeof(float), hipMemcpyDeviceToHost);

            // Apply GPU Adam optimization (no need to download gradients)
            gaussians.adam_step_gpu(learning_rate, 0.9f, 0.999f, 1e-8f, iteration + 1);
            
            auto end_time = std::chrono::high_resolution_clock::now();
            auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
            
            // Print progress
            if (iteration % 10 == 0) {
                std::cout << "Iteration " << std::setw(4) << iteration 
                          << " | Loss: " << std::scientific << std::setprecision(6) << total_loss
                          << " | Time: " << duration.count() << "ms" << std::endl;
            }
            
            // Save intermediate results
            if (iteration % save_interval == 0) {
                save_current_rendering(iteration);
            }
        }
        
        // Save final result
        save_current_rendering(max_iterations);
        
        std::cout << "Training completed!" << std::endl;
    }
};

int main(int argc, char** argv) {
    std::cout << "Gaussian Splatting Training with CUDA Automatic Differentiation" << std::endl;
    std::cout << "================================================================" << std::endl;
    
    // Check CUDA availability
    int device_count;
    hipGetDeviceCount(&device_count);
    if (device_count == 0) {
        std::cerr << "No CUDA devices available!" << std::endl;
        return -1;
    }
    std::cout << "Found " << device_count << " CUDA device(s)" << std::endl;
    
    // Set device
    hipSetDevice(0);
    
    // Create trainer
    GaussianSplattingTrainer trainer(0.01f, 500, 25);  // lr=0.01, max_iter=500, save every 25 iterations
    
    // Load target image
    std::string image_file = "data/target.png";
    if (argc > 1) {
        image_file = argv[1];
    }
    
    if (!trainer.load_target_image(image_file)) {
        std::cerr << "Failed to load target image" << std::endl;
        return -1;
    }
    
    // Initialize Gaussians
    trainer.initialize_gaussians();
    
    // Start training
    try {
        trainer.train();
    } catch (const std::exception& e) {
        std::cerr << "Training failed with exception: " << e.what() << std::endl;
        return -1;
    }
    
    std::cout << "Program completed successfully!" << std::endl;
    return 0;
}