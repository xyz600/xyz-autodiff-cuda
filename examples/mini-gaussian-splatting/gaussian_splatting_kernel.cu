#include "hip/hip_runtime.h"
#include "gaussian_splatting_kernel.cuh"
#include <iostream>

__global__ void gaussian_splatting_kernel(
    const GaussianParams* gaussians,
    GaussianGrads* gradients, 
    const float* target_image,
    PixelOutput* output_image,
    int image_width,
    int image_height,
    int num_gaussians
) {
    // Calculate pixel coordinates from block and thread indices
    int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pixel_y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check bounds
    if (pixel_x >= image_width || pixel_y >= image_height) return;
    
    int pixel_idx = pixel_y * image_width + pixel_x;
    
    // Initialize output pixel
    PixelOutput& pixel_out = output_image[pixel_idx];
    pixel_out.color[0] = 0.0f;
    pixel_out.color[1] = 0.0f;
    pixel_out.color[2] = 0.0f;
    pixel_out.alpha = 0.0f;
    pixel_out.loss = 0.0f;
    
    // Query point (current pixel position)
    const float query_point[2] = {static_cast<float>(pixel_x), static_cast<float>(pixel_y)};
    
    // Accumulate total color from all Gaussians
    Variable<3, float> total_color;
    total_color[0] = 0.0f;
    total_color[1] = 0.0f;
    total_color[2] = 0.0f;
    
    for (int g = 0; g < num_gaussians; g++) {
        const GaussianParams& gauss = gaussians[g];
        
        // Create Variable references for this Gaussian's parameters (no gradients yet)
        VariableRef<2, float> center(const_cast<float*>(gauss.center), nullptr);
        VariableRef<2, float> scale(const_cast<float*>(gauss.scale), nullptr);
        VariableRef<1, float> rotation(const_cast<float*>(gauss.rotation), nullptr);
        VariableRef<3, float> color(const_cast<float*>(gauss.color), nullptr);
        VariableRef<1, float> opacity(const_cast<float*>(gauss.opacity), nullptr);
        
        // Build computation graph
        auto covariance = op::scale_rotation_to_covariance_3param(scale, rotation);
        auto inv_covariance = op::sym_matrix2_inv(covariance);
        auto mahalanobis_dist_sq = op::mahalanobis_distance_with_center(query_point[0], query_point[1], center, inv_covariance);
        auto scaled_distance = mahalanobis_dist_sq * 0.5f;
        auto neg_scaled = op::neg(scaled_distance);
        auto gaussian_value = op::exp(neg_scaled);
        auto weighted_gauss = gaussian_value * opacity;
        auto gauss_broadcast = op::broadcast<3>(weighted_gauss);
        auto weighted_color = color * gauss_broadcast;

        // Run forward pass only
        weighted_color.forward();

        // Accumulate to total color
        total_color[0] += weighted_color[0];
        total_color[1] += weighted_color[1];
        total_color[2] += weighted_color[2];
    }
    
    // Set output pixel color for visualization
    pixel_out.color[0] = total_color[0];
    pixel_out.color[1] = total_color[1];
    pixel_out.color[2] = total_color[2];
    
    // Create target image variable (constant)
    int target_idx = pixel_idx * 3;
    Variable<3, float> target_color;
    target_color[0] = target_image[target_idx + 0];
    target_color[1] = target_image[target_idx + 1];
    target_color[2] = target_image[target_idx + 2];
    
    // Compute gradients for each Gaussian using L1 norm automatic differentiation
    for (int g = 0; g < num_gaussians; g++) {
        const GaussianParams& gauss = gaussians[g];
        GaussianGrads& grads = gradients[g];
        
        // Create Variable references with local gradient buffers
        // Create Variable references for this Gaussian's parameters (no gradients yet)
        VariableRef<2, float> center(const_cast<float*>(gauss.center), grads.center);
        VariableRef<2, float> scale(const_cast<float*>(gauss.scale), grads.scale);
        VariableRef<1, float> rotation(const_cast<float*>(gauss.rotation), grads.rotation);
        VariableRef<3, float> color(const_cast<float*>(gauss.color), grads.color);
        VariableRef<1, float> opacity(const_cast<float*>(gauss.opacity), grads.opacity);
        
        // Build computation graph for this Gaussian
        auto covariance = op::scale_rotation_to_covariance_3param(scale, rotation);
        auto inv_covariance = op::sym_matrix2_inv(covariance);
        auto mahalanobis_dist_sq = op::mahalanobis_distance_with_center(query_point[0], query_point[1], center, inv_covariance);
        auto scaled_distance = mahalanobis_dist_sq * 0.5f;
        auto neg_scaled = op::neg(scaled_distance);
        auto gaussian_value = op::exp(neg_scaled);
        auto weighted_gauss = gaussian_value * opacity;
        auto gauss_broadcast = op::broadcast<3>(weighted_gauss);
        auto weighted_color = color * gauss_broadcast;
        
        // Build full L1 loss computation graph for this Gaussian
        auto color_diff = op::sub(weighted_color, target_color);
        auto l1_loss = op::l1_norm(color_diff);
        
        // Run complete forward and backward pass
        l1_loss.run();
    }
}

void launch_gaussian_splatting(
    const GaussianParams* device_gaussians,
    GaussianGrads* device_gradients,
    const float* device_target_image, 
    PixelOutput* device_output_image,
    int image_width,
    int image_height,
    int num_gaussians
) {
    // Calculate grid dimensions for 16x16 tiles
    dim3 block_size(TILE_SIZE, TILE_SIZE);
    dim3 grid_size(
        (image_width + TILE_SIZE - 1) / TILE_SIZE,
        (image_height + TILE_SIZE - 1) / TILE_SIZE
    );
    
    // Launch kernel
    gaussian_splatting_kernel<<<grid_size, block_size>>>(
        device_gaussians,
        device_gradients,
        device_target_image,
        device_output_image,
        image_width,
        image_height,
        num_gaussians
    );
    
    // Check for kernel launch errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch error: " << hipGetErrorString(err) << std::endl;
        return;
    }
    
    // Wait for kernel completion
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Kernel execution error: " << hipGetErrorString(err) << std::endl;
        return;
    }
}
